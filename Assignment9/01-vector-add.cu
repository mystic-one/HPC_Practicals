
#include <hip/hip_runtime.h>
#include <stdio.h>

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
  // calculating the thread mapping and strides
  int current = threadIdx.x + blockIdx.x * blockDim.x;
  int gridStride = gridDim.x * blockDim.x;
  for(int i = current; i < N; i += gridStride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  // allocating the memory using cudaMallocManaged
  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);
   
  // fixing number of threads in a block
  int no_of_threads = 256;
  // calculating number of blocks required
  int no_of_blocks = N-1+no_of_threads/no_of_threads;

  // invoking the kernel with specific environment configuration
  addVectorsInto<<<no_of_blocks, no_of_threads>>>(c, a, b, N);
  
  // synchronizing GPU and CPU
  hipDeviceSynchronize();
    
  checkElementsAre(7, c, N);

  // deallocating the memory using cudaFree()
  hipFree(a);
  hipFree(b);
  hipFree(c);
}
